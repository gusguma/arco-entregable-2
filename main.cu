#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////
/// PROGRAMACIÓN EN CUDA C/C++
/// Práctica:	ENTREGABLE 2 : Gráficos en CUDA
/// Autor:		Angel Sierra López, Gustavo Gutiérrez Martín
/// Fecha:		Noviembre 2022
///////////////////////////////////////////////////////////////////////////

/// Dependencias
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "gpu_bitmap.h"
#include <ctime>

/// Constantes
#define MB (1<<20) /// MiB = 2^20
/// Dimension del bitmap horizontal
#define WIDTH 512
/// Dimension del bitmap vertical
#define HEIGHT 512
/// Numero de hilos
#define THREADS 16
/// Definimos el número de celdas en horizontal
#define CELLS_WIDTH 8
/// Definimos el número de celdas en vertical
#define CELLS_HEIGHT 8

/// Funciones
/// numero de CUDA cores
int getCudaCores(hipDeviceProp_t deviceProperties);
/// GLOBAL: funcion llamada desde el host y ejecutada en el device (kernel)
__global__ void kernel( unsigned char *imagen );

/// MAIN: rutina principal ejecutada en el host
int main() {
    /// almacena el número de devices disponibles
    int deviceCount;
    /// buscando dispositivos
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        /// mostramos el error si no se encuentra un dispositivo
        printf("¡No se ha encontrado un dispositivo CUDA!\n");
        printf("<pulsa [INTRO] para finalizar>");
        getchar();
        return 1;
    } else {
        ///obtenemos las propiedades del dispositivo CUDA
        int deviceId = 0;
        hipDeviceProp_t deviceProp{};
        hipGetDeviceProperties(&deviceProp, deviceId);
        int SM = deviceProp.multiProcessorCount;
        int cudaCores = getCudaCores(deviceProp);
        printf("***************************************************\n");
        printf("DEVICE: %s\n", deviceProp.name);
        printf("***************************************************\n");
        printf("- Capacidad de Computo            \t: %d.%d\n", deviceProp.major, deviceProp.minor);
        printf("- No. de MultiProcesadores        \t: %d \n", SM);
        printf("- No. de CUDA Cores (%dx%d)       \t: %d \n", cudaCores, SM, cudaCores * SM);
        printf("- Memoria Global (total)          \t: %zu MiB\n", deviceProp.totalGlobalMem / MB);
        printf("- No. maximo de Hilos (por bloque)\t: %d\n", deviceProp.maxThreadsPerBlock);
        printf("***************************************************\n");
    }
    /// declaracion de eventos
    hipEvent_t start;
    hipEvent_t stop;
    /// Declaracion del bitmap:
    /// Inicializacion de la estructura RenderGPU
    RenderGPU foto(WIDTH, HEIGHT);
    /// Obtenemos el tamaño del bitmap en bytes
    size_t size = foto.image_size();
    /// Asignacion y reserva de la memoria en el host (framebuffer)
    unsigned char *host_bitmap = foto.get_ptr();
    /// Reserva de memoria en el device
    unsigned char *dev_bitmap;
    hipMalloc( (void**)&dev_bitmap, size );
    /// Lanzamos un kernel bidimensional con bloques de 256 hilos (16x16)
    dim3 hilosB(THREADS,THREADS);
    /// Calculamos el numero de bloques necesario (un hilo por cada pixel)
    dim3 Nbloques(WIDTH/THREADS, HEIGHT/THREADS);
    /// creacion de eventos para calcular el tiempo de GPU
    hipEventCreate(&start);
    hipEventCreate(&stop);
    /// marca inicio proceso GPU
    hipEventRecord(start,0);
    /// Generamos el bitmap
    kernel<<<Nbloques,hilosB>>>( dev_bitmap );
    /// marca final proceso GPU
    hipEventRecord(stop,0);
    /// sincronizacion GPU-CPU
    hipEventSynchronize(stop);
    /// cálculo del tiempo de GPU en milisegundos
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime,start,stop);
    /// impresion de resultados
    printf("> Tiempo de ejecucion GPU: %f ms\n",elapsedTime);
    printf("***************************************************\n");
    /// Copiamos los datos desde la GPU hasta el framebuffer para visualizarlos
    hipMemcpy( host_bitmap, dev_bitmap, size, hipMemcpyDeviceToHost );
    /// liberacion de recursos
    hipEventDestroy(start);
    hipEventDestroy(stop);
    /// función que muestra por pantalla la salida del programa
    time_t fecha;
    time(&fecha);
    printf("> Programa ejecutado el: %s", ctime(&fecha));
    printf("***************************************************\n");
    /// capturamos un INTRO para que no se cierre la consola de MSVS
    /// Visualizacion y salida
    printf("\n...pulsa [ESC] para finalizar...");
    foto.display_and_exit();
    return 0;
}

int getCudaCores(hipDeviceProp_t deviceProperties) {
    int cudaCores = 0;
    int major = deviceProperties.major;
    if (major == 1) {
        /// TESLA
        cudaCores = 8;
    } else if (major == 2) {
        /// FERMI
        if (deviceProperties.minor == 0) {
            cudaCores = 32;
        } else {
            cudaCores = 48;
        }
    } else if (major == 3) {
        /// KEPLER
        cudaCores = 192;
    } else if (major == 5) {
        /// MAXWELL
        cudaCores = 128;
    } else if (major == 6 || major == 7 || major == 8) {
        /// PASCAL, VOLTA (7.0), TURING (7.5), AMPERE
        cudaCores = 64;
    } else {
        /// ARQUITECTURA DESCONOCIDA
        cudaCores = 0;
        printf("¡Dispositivo desconocido!\n");
    }
    return cudaCores;
}

__global__ void kernel( unsigned char *imagen ) {
    /// Kernel bidimensional multibloque
    /// coordenada horizontal de cada hilo
    unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
    /// coordenada vertical de cada hilo
    unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;
    /// indice global de cada hilo (indice lineal para acceder a la memoria)
    unsigned int myID = x + y * blockDim.x * gridDim.x;
    /// cada hilo obtiene la posicion de su pixel
    unsigned int miPixel = myID * 4;
    /// calculamos en que posicion x dentro del tablero de 8x8 esta el pixel
    unsigned int positionX = (x * CELLS_WIDTH) / WIDTH;
    /// calculamos en que posicion y dentro del tablero de 8x8 esta el pixel
    unsigned int positionY = (y * CELLS_HEIGHT) / HEIGHT;
    /// cada hilo rellena los 4 canales de su pixel, si la positionX + la positionY es par, rellena de blanco, sino negro
    if ((positionX + positionY) % 2) {
        /// rellena el pixel de color blanco
        imagen[miPixel] = 255; /// canal R
        imagen[miPixel + 1] = 255;/// canal G
        imagen[miPixel + 2] = 255; /// canal B
        imagen[miPixel + 3] = 0; /// canal alfa
    } else {
        /// rellena el pixel de color negro
        imagen[miPixel] = 0; /// canal R
        imagen[miPixel + 1] = 0;/// canal G
        imagen[miPixel + 2] = 0; /// canal B
        imagen[miPixel + 3] = 0; /// canal alfa
    }
}
